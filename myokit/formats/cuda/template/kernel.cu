#include "hip/hip_runtime.h"
<?
#
# kernel.cu
#
# A pype template for a CUDA kernel
#
# Required variables
#-------------------
# model    A model
# use_rl   A boolean (use Rush-Larsen yes/no)
#
# This file is part of Myokit.
# See http://myokit.org for copyright, sharing, and licensing details.
#
# Authors:
#  Michael Clerx
#  Enno de Lange
#
import myokit
import myokit.formats.cuda as cuda

# Double or single precision?
precision = myokit.SINGLE_PRECISION

# Check if model has diffusion_current binding
if model.binding('diffusion_current') is None:
    raise ValueError('Model should have a variable with binding `diffusion_current`.')

# Clone model, and adapt to inf-tau form if in RL mode
rl_states = {}
if use_rl:
    # Check vm is known
    if model.label('membrane_potential') is None:
        raise ValueError('Model should have a variable with label `membrane_potential`.')

    # Convert model to inf-tau form (returns clone) and get vm
    import myokit.lib.hh as hh
    model = hh.convert_hh_states_to_inf_tau_form(model)
    vm = model.label('membrane_potential')

    # Get (inf, tau) tuple for every Rush-Larsen state
    for state in model.states():
        res = hh.get_inf_and_tau(state, vm)
        if res is not None:
            rl_states[state] = res

else:
    # Clone model
    model = model.clone()

# Merge interdepdent components
model.resolve_interdependent_components()

# Reserve keywords
model.reserve_unique_names(*cuda.keywords)
model.reserve_unique_names(
    *['calc_' + c.name() for c in model.components()]
    )
model.reserve_unique_names(
    #'time',
    #'pace',
    'I_diff',
    'dt',
    'parameters',
    'state',
    )
model.create_unique_names()

# Get initial values (before time binding is removed and model becomes invalid)
initial_values = model.initial_values(True)

# Process bindings, remove unsupported bindings, get map of bound variables to
# internal names
bound_variables = myokit._prepare_bindings(model, {
    #'time' : 'time',
    #'pace' : 'pace',
    'diffusion_current' : 'I_diff',
})

# Get equations
equations = model.solvable_order()

# Delete "*remaning*" group, guaranteed to be empty with independent components
del(equations['*remaining*'])

# Get component order
comp_order = equations.keys()
comp_order = [model.get(c) for c in comp_order]

# Get component inputs/output arguments
comp_in, comp_out = model.map_component_io(
    omit_states=True,
    omit_derivatives=False,
    omit_constants=True,
    rl_states=rl_states,
)

# Bound variables will be passed in to every function as needed, so they can be
# removed from the input/output lists
def clear_io_list(comp_list):
    for comp, clist in comp_list.items():
        for var in bound_variables:
            lhs = var.lhs()
            while lhs in clist:
                clist.remove(lhs)
clear_io_list(comp_in)
clear_io_list(comp_out)

# Components that use one of the bound variables should get it as an input
# variable.
for comp, clist in comp_in.items():
    for bound in bound_variables:
        lhs = bound.lhs()
        if lhs in clist:
            continue
        for var in comp.variables(deep=True):
            if var.rhs().depends_on(lhs):
                clist.append(lhs)
                break

# Get expression writer
w = cuda.CudaExpressionWriter(precision=precision)

# Define var/lhs function
def v(var):
    """
    Accepts a variable or a left-hand-side expression and returns its C
    representation.
    """
    if isinstance(var, myokit.Derivative):
        # Explicitly asked for derivative
        return 'D_' + var.var().uname()
    if isinstance(var, myokit.Name):
        var = var.var()
    if var in bound_variables:
        return bound_variables[var]
    return var.uname()
w.set_lhs_function(v)

# Tab
tab = '    '

# To render last function inline, set "last_component" to the last component
#last_component = None
last_component = comp_order[-1]


export = 'CUDA export' + (' with RL updates' if use_rl else '')
print('/*')
print(' * CUDA kernel for ' + model.name())
print(' *')
print(' * Generated on ' + myokit.date() + ' by Myokit ' + export)
print(' */')


if precision == myokit.SINGLE_PRECISION:
    print('#include <float.h>')

?>

////////////////////////////////////////////////////////////////////////////////
// Macros and definitions
////////////////////////////////////////////////////////////////////////////////

<?
if precision == myokit.SINGLE_PRECISION:
    print('/* Using single precision floats */')
    print('typedef float Real;')
else:
    print('/* Using double precision floats */')
    print('typedef double Real;')
?>

#define NDIM <?=str(model.count_states())?>

/* Accessor macros */
#define N_FREE_PARAMETERS 0

<?
print('/* Constants */')
for group in equations.values():
    for eq in group.equations(const=True):
        if isinstance(eq.rhs, myokit.Number):
            print('#define ' + v(eq.lhs) + ' ' + w.ex(eq.rhs))

print('')
print('/* Calculated constants */')
for group in equations.values():
    for eq in group.equations(const=True):
        if not isinstance(eq.rhs, myokit.Number):
            print('#define ' + v(eq.lhs) + ' (' + w.ex(eq.rhs) + ')')

print('')
print('/* Aliases of state variables. */')
for var in model.states():
    print('#define ' + var.uname() + ' state[' + str(var.index()) + ']')

?>

////////////////////////////////////////////////////////////////////////////////
// Local function declarations
////////////////////////////////////////////////////////////////////////////////

<?
print('')
print('/* Components */')
for comp, ilist in comp_in.items():
    if comp == last_component:
        continue
    olist = comp_out[comp]
    if len(olist) == 0:
        continue

    # Function header
    args = ['Real *state']
    args.extend(['Real '  + v(lhs) for lhs in ilist])
    args.extend(['Real& ' + v(lhs) for lhs in olist])
    name = 'calc_' + comp.name()
    print('__device__ void ' + name + '(' + ', '.join(args) + ')')
    print('{')

    # Equations
    for eq in equations[comp.name()].equations(const=False):
        var = eq.lhs.var()
        pre = tab
        if not (eq.lhs in ilist or eq.lhs in olist):
            if var in rl_states:
                continue
            pre += 'Real '
        if var not in bound_variables:
            print(pre + w.eq(eq) + ';')

    print('}')
    print('')
?>

////////////////////////////////////////////////////////////////////////////////
//! Compute an Euler step of the model.
////////////////////////////////////////////////////////////////////////////////
__device__ int iterate_euler_cu(const Real dt, Real *state, Real I_diff,
                                    Real *parameters)
{
<?
print(tab + '/* Evaluate derivatives */')
for comp in comp_order:
    ilist = comp_in[comp]
    olist = comp_out[comp]

    # Skip components without output
    if len(olist) == 0:
        continue

    # Skip last component (if in inline mode)
    if comp == last_component:
        continue

    # Declare any output variables
    for var in comp_out[comp]:
        print(tab + 'Real ' + v(var) + ' = 0;')

    # Function header
    args = ['state']
    args.extend([v(lhs) for lhs in ilist])
    args.extend([v(lhs) for lhs in olist])
    print(tab + 'calc_' + comp.name() + '(' + ', '.join(args) + ');')

if last_component:
    print(tab)
    print(tab + '/* Evaluate ' + last_component.name() + ' */')
    olist = comp_out[last_component]
    ilist = comp_in[last_component]
    for eq in equations[last_component.name()].equations(const=False):
        var = eq.lhs.var()
        pre = tab
        if not eq.lhs in ilist:
            pre += 'Real '
        if var not in bound_variables:
            print(pre + w.eq(eq) + ';')

?>
    /* Perform update */
<?
for var in model.states():
    if var in rl_states:
        inf, tau = rl_states[var]
        inf, tau, var = v(inf), v(tau), v(var)
        print(tab + var + ' = ' + inf + ' - (' + inf + ' - ' + var + ') * exp(-dt / ' + tau + ');')
    else:
        print(tab + v(var) + ' += dt * ' + v(var.lhs()) + ';')
?>

    return 0;
}

/* Set the standard initial conditions */
int get_default_initial_state(Real *state)
{
    if (state == 0) return(-1);

<?
for var, val in zip(model.states(), initial_values):
    if 'desc' in var.meta:
        print(tab + '// ' + var.meta['desc'])
    print(tab + v(var) + ' = ' + myokit.float.str(val) + ';')
?>

    return(0);
}

/* Function to initialize the parameter array in the model structure. */
int get_default_parameters(Real *parameters)
{
    if (parameters == NULL) return(1);
    return(0);
}

<?
print('/* Remove constant definitions */')
for group in equations.values():
    for eq in group.equations(const=True):
        print('#undef ' + v(eq.lhs))

print('')
print('/* Remove aliases of state variables. */')
for var in model.states():
    print('#undef ' + var.uname())

?>
